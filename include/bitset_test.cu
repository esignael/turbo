#include "hip/hip_runtime.h"
#include "bitset.cuh"
#include <chrono>
typedef std::chrono::high_resolution_clock Clock;

template <size_t array_size>
__global__ void kern (Bitset<array_size> &a) {
   a.add(1);
   a.add(-1);
   a.add(-2);
   a.add(0);
   a.add(-160);
   a.add(160);
   a.add(-394);
   a.remove(-1);
   a.remove(-394);
}

template <size_t array_size>
__global__ void diff_test (Bitset<array_size> &a, Bitset<array_size> &b) {
}

int main () {
   Bitset<20> *a = new Bitset<20>;
   Bitset<20> *b = new Bitset<20>;
   for(int i=31;i<400;++i){a->add(i); }
   for(int i=-100;i<-32;++i){b->add(i); }
   /*
   a.add(1);
   a.add(-1);
   a.add(-2);
   a.add(0);
   a.add(-160);
   a.print();
   a.add(160);
   a.add(-394);
   a.remove(-1);
   a.remove(-394);
   */

   hipDeviceSynchronize();
   kern<<<1,20>>>(*a);
   hipDeviceSynchronize();
   b->print();

   hipDeviceSynchronize();
   diff_test<<<1,42>>>(*b, *a);
   hipDeviceSynchronize();
   printf("=========== B ============\n");
   b->print();
   printf("=========== A ============\n");
   a->print();
   printf("=========== B / A============\n");
   (b->diff(*a)).print();
   Bitset<20> c;
   c = b->diff(*a);
   printf("===========   C  ============\n");
   c.add(-32);
   c.print();
   c.max();
   printf("min: %i\n", a->min());
   
   return 0;
};
